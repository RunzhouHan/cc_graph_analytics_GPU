#include "hip/hip_runtime.h"
/**
 File name: bfs_gpu.cu
 Author: Yuede Ji
 Date: 
    from 03-02-2016 2:36
**/
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <set>
#include <string.h>
#include <queue>
//#include "translator_json_csr.h"
#include "wtime.h"
#include "graph.h"

#define THREADS_PER_BLOCK 256
#define BLOCKS_PER_GRID 32768
#define STAND_THREADS 256
#define STAND_BLOCKS 256
#define THREAD_BIN_SIZE 32

#define INF (int)(1<<30)

#define visited_color 1
#define unvisited_color 0
#define frontier_color 2

typedef int bit_type;

const char output_file[] = "bfs_result.txt";
const char time_detail[] = "time_detail.csv";


int pivot_selection_first(
        graph *g, 
        index_t vertex_count)
{
    long int max_out_degree = 0;
    index_t index = 0;
    for(index_t j=0; j<vertex_count; ++j)
    {
        long int temp_out_degree = g->beg_pos[j+1] - g->beg_pos[j];
        if(temp_out_degree > max_out_degree)
        {
            max_out_degree = temp_out_degree;
            index = j;
        }
    }

    printf("first_pivot = %d, out_degree = %d\n", index, max_out_degree);
    return index;
}

__global__ void bfs_sync_color_top_down_first(
        index_t *d_adj_list, 
        index_t *d_beg_pos, 
        bit_type *d_vertex_status, 
        bool *d_change, 
        index_t *d_vertex_count,
        index_t *d_level,
        index_t *d_frontier_queue,
        index_t *d_thread_bin)
{
    index_t id = blockIdx.x*blockDim.x + threadIdx.x;
    const index_t THDS_COUNT=blockDim.x*gridDim.x;
    index_t vertex_count = *d_vertex_count;
    index_t level = *d_level;
//    index_t bin_offset = 0;
//    index_t id = tid;
//    index_t begin_offset = 0;
//    if(id != 0)
//        begin_offset = tid * THREAD_BIN_SIZE;
    while(id < vertex_count)
    {
        if(d_vertex_status[id] == level)
        {
            for(index_t i=d_beg_pos[id]; i<d_beg_pos[id+1]; ++i)
            {
                index_t w = d_adj_list[i];

                ///atomic operation to guarantee only thread has w
                if(d_vertex_status[w] == unvisited_color)
                {
                    d_vertex_status[w] = level + 1;
                    *d_change = true;
//                    d_thread_bin[bin_offset] = w;
//                    bin_offset ++;
                }
            }
        }
        id += THDS_COUNT;
    }
}

__global__ void bfs_bottom_up_first(
        bit_type *d_vertex_status_fw, 
        index_t *d_adj_list_bw, 
        index_t *d_beg_pos_bw, 
        bool *d_change, 
        bool *d_mark, 
        index_t *d_vertex_count,
        index_t *d_level)
{
    index_t id = blockIdx.x*blockDim.x + threadIdx.x;
    const index_t THDS_COUNT=blockDim.x*gridDim.x;
    index_t vertex_count = *d_vertex_count;
    while(id < vertex_count)
    {
        if(!d_mark[id] && d_vertex_status_fw[id] == 0)
        {
            for(index_t i=d_beg_pos_bw[id]; i<d_beg_pos_bw[id+1]; ++i)
            {
                index_t w = d_adj_list_bw[i];
                if(!d_mark[w] && (d_vertex_status_fw[w] != unvisited_color))
                {
                    d_vertex_status_fw[id] = *d_level;
                    *d_change = true;
                    break;
                   // printf("%d\n", id);
                }
            }
        }
        id += THDS_COUNT;
    }
}

int main(int args, char **argv)
{
    srand(time(NULL));
	hipSetDevice(0);
    printf("args = %d\n", args);
    if(args != 4)
    {
        printf("Usage: ./scc <fw_beg_filename> <fw_csr_filename> <source>\n");
        exit(-1);
    }

    char *fw_beg_filename = argv[1];
    char *fw_csr_filename = argv[2];
    index_t root = atoi(argv[3]);

    graph *g = new graph(fw_beg_filename, fw_csr_filename);
    const index_t vertex_count = g->vert_count + 1;
    const index_t edge_count = g->edge_count;
    
	bit_type *forward_vertex_status = (bit_type *)malloc(sizeof(bit_type)*vertex_count);
    index_t *frontier_queue = (index_t*)malloc(sizeof(index_t)*vertex_count);
	   
	//------------------------------------------------------------------------
    //Deciding how many blocks to be used   
    index_t number_of_blocks = 1;
    index_t number_of_threads_per_block = vertex_count;
     
    if(vertex_count > THREADS_PER_BLOCK)
    {
        number_of_blocks = (index_t)ceil(vertex_count/(double)THREADS_PER_BLOCK);
        number_of_threads_per_block = THREADS_PER_BLOCK;
        if(number_of_blocks > BLOCKS_PER_GRID)
            number_of_blocks = BLOCKS_PER_GRID;
    }
    
//    printf("blocks = %d, threads = %d\n", number_of_blocks, number_of_threads_per_block);
    //------------------------------------------------------------------------
    //allocating auxiliars in CPU
    for(index_t i = 0 ; i < vertex_count ; ++i)
    {
        forward_vertex_status[i] = 0;//no colors 
    }
    //index_t root = pivot_selection_first(g, vertex_count);
    forward_vertex_status[root] = 1;
    //------------------------------------------------------------------------
    //Allocating GPU memory:

    index_t *d_adj_list_direct;
    hipMalloc((void**) &d_adj_list_direct, sizeof(index_t)*edge_count);
    hipMemcpy( d_adj_list_direct, g->csr, sizeof(index_t)*edge_count, hipMemcpyHostToDevice);

	index_t *d_beg_pos_direct;
    hipMalloc((void**) &d_beg_pos_direct, sizeof(index_t)*(vertex_count + 1));
    hipMemcpy( d_beg_pos_direct, g->beg_pos, sizeof(index_t)*(vertex_count + 1), hipMemcpyHostToDevice);

	bit_type *d_forward_vertex_status;
	hipMalloc((void**) &d_forward_vertex_status, sizeof(bit_type)*vertex_count);
    hipMemcpy( d_forward_vertex_status, forward_vertex_status, sizeof(index_t)*vertex_count, hipMemcpyHostToDevice);

    index_t *d_vertex_count;
    hipMalloc((void **) &d_vertex_count, sizeof(index_t));
    hipMemcpy(d_vertex_count, &(vertex_count), sizeof(index_t), hipMemcpyHostToDevice);
    
    index_t offset = 0;
    index_t *d_offset;
    hipMalloc((void **) &d_offset, sizeof(index_t));
    hipMemcpy(d_offset, &(offset), sizeof(index_t), hipMemcpyHostToDevice);

    index_t *d_frontier_queue;
    hipMalloc((void **) &d_frontier_queue, sizeof(index_t) * vertex_count);
    
    index_t *d_thread_bin;
    hipMalloc((void **) &d_thread_bin, sizeof(index_t) * STAND_BLOCKS * STAND_THREADS * THREAD_BIN_SIZE);
    hipMemcpy(d_vertex_count, &(vertex_count), sizeof(index_t), hipMemcpyHostToDevice);
    //CPU & GPU shared variable
    
    index_t * level;
    index_t * d_level;
    hipHostAlloc((void **) &level, sizeof(index_t), hipHostMallocMapped);
    hipHostGetDevicePointer(&d_level, level, 0);
    *level = 0;
    
    bool * change;
    bool * d_change;
    hipHostAlloc((void **) &change, sizeof(bool), hipHostMallocMapped);
    hipHostGetDevicePointer(&d_change, change, 0);
    *change = true;

    FILE * fp_time = fopen(time_detail, "w");
    double time = wtime();
    while(*change)
    {
        (*level) ++;
        *change = false;
        double temp_time_beg = wtime();

        bfs_sync_color_top_down_first<<<STAND_BLOCKS, STAND_THREADS>>>(
                d_adj_list_direct, 
                d_beg_pos_direct, 
                d_forward_vertex_status, 
                d_change, 
                d_vertex_count,
                d_level,
                d_frontier_queue,
                d_thread_bin);
        hipDeviceSynchronize();
        double temp_time = wtime() - temp_time_beg;
        fprintf(fp_time, "%d, %g\n", *level, temp_time*1000);
    }
    double bfs_fw_time = wtime() - time;
    fclose(fp_time);
    printf("depth = %d\n", *level);
    printf("bfs fw time = %g (ms)\n", bfs_fw_time * 1000);
    hipMemcpy(forward_vertex_status, d_forward_vertex_status, sizeof(index_t)*vertex_count, hipMemcpyDeviceToHost);
    FILE * fp_out = fopen(output_file, "w");
    for(index_t i=0; i<vertex_count; ++i)
    {
        fprintf(fp_out, "%d %d\n", i, forward_vertex_status[i]);
    }
    fclose(fp_out);

	free(forward_vertex_status);
    hipFree(d_adj_list_direct);
    hipFree(d_beg_pos_direct);
	hipFree(d_forward_vertex_status);
	hipFree(d_change);
	return 0;
}

